#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "rasterizer_impl.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

#include "auxiliary.h"
#include "forward.h"
#include "backward.h"

// Helper function to find the next-highest bit of the MSB
// on the CPU.
uint32_t getHigherMsb(uint32_t n)
{
	uint32_t msb = sizeof(n) * 4;
	uint32_t step = msb;
	while (step > 1)
	{
		step /= 2;
		if (n >> msb)
			msb += step;
		else
			msb -= step;
	}
	if (n >> msb)
		msb++;
	return msb;
}

// Wrapper method to call auxiliary coarse frustum containment test.
// Mark all Gaussians that pass it.
__global__ void checkFrustum(int P,
							 const float *orig_points,
							 const float *viewmatrix,
							 const float *projmatrix,
							 bool *present)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	float3 p_view;
	present[idx] = in_frustum(idx, orig_points, viewmatrix, projmatrix, false, p_view);
}

// Generates one key/value pair for all Gaussian / tile overlaps.
// Run once per Gaussian (1:N mapping).
__global__ void duplicateWithKeys(
	int P,
	const float2 *points_xy,
	const float *depths,
	const uint32_t *offsets,
	uint64_t *gaussian_keys_unsorted,
	uint32_t *gaussian_values_unsorted,
	int *radii,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Gaussians
	if (radii[idx] > 0)
	{
		// Find this Gaussian's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];
		uint2 rect_min, rect_max;

		getRect(points_xy[idx], radii[idx], rect_min, rect_max, grid);

		// For each tile that the bounding rect overlaps, emit a
		// key/value pair. The key is |  tile ID  |      depth      |,
		// and the value is the ID of the Gaussian. Sorting the values
		// with this key yields Gaussian IDs in a list, such that they
		// are first sorted by tile and then by depth.
		for (int y = rect_min.y; y < rect_max.y; y++)
		{
			for (int x = rect_min.x; x < rect_max.x; x++)
			{
				uint64_t key = y * grid.x + x;
				key <<= 32;
				key |= *((uint32_t *)&depths[idx]);
				gaussian_keys_unsorted[off] = key;
				gaussian_values_unsorted[off] = idx;
				off++;
			}
		}
	}
}

__global__ void createWithKeys(
	int P,
	const float2 *points_xy,
	const float *depths,
	const uint32_t *offsets,
	const uint32_t *tiles_touched,
	uint64_t *points_keys_unsorted,
	uint32_t *points_values_unsorted,
	dim3 grid)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Generate no key/value pair for invisible Points
	if (tiles_touched[idx] > 0)
	{
		// Find this Point's offset in buffer for writing keys/values.
		uint32_t off = (idx == 0) ? 0 : offsets[idx - 1];

		// determine the tile that the point is in
		const float2 p = points_xy[idx];
		int x = min(grid.x - 1, max((int)0, (int)(p.x / BLOCK_X)));
		int y = min(grid.y - 1, max((int)0, (int)(p.y / BLOCK_Y)));

		uint64_t key = y * grid.x + x;
		key <<= 32;
		key |= *((uint32_t *)&depths[idx]);
		points_keys_unsorted[off] = key;
		points_values_unsorted[off] = idx;
	}
}

// Check keys to see if it is at the start/end of one tile's range in
// the full sorted list. If yes, write start/end of this tile.
// Run once per instanced (duplicated) Gaussian ID.
__global__ void identifyTileRanges(int L, uint64_t *point_list_keys, uint2 *ranges)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];
	uint32_t currtile = key >> 32;
	if (idx == 0)
		ranges[currtile].x = 0;
	else
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32;
		if (currtile != prevtile)
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

CudaRasterizer::GeometryState CudaRasterizer::GeometryState::fromChunk(char *&chunk, size_t P)
{
	GeometryState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.clamped, P * 3, 128);
	obtain(chunk, geom.internal_radii, P, 128);
	obtain(chunk, geom.means2D, P, 128);
	obtain(chunk, geom.cov3D, P * 6, 128);
	obtain(chunk, geom.view2gaussian, P * 16, 128); // could be 12
	obtain(chunk, geom.conic_opacity, P, 128);
	obtain(chunk, geom.rgb, P * 3, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::PointState CudaRasterizer::PointState::fromChunk(char *&chunk, size_t P)
{
	PointState geom;
	obtain(chunk, geom.depths, P, 128);
	obtain(chunk, geom.points2D, P, 128);
	obtain(chunk, geom.tiles_touched, P, 128);
	hipcub::DeviceScan::InclusiveSum(nullptr, geom.scan_size, geom.tiles_touched, geom.tiles_touched, P);
	obtain(chunk, geom.scanning_space, geom.scan_size, 128);
	obtain(chunk, geom.point_offsets, P, 128);
	return geom;
}

CudaRasterizer::ImageState CudaRasterizer::ImageState::fromChunk(char *&chunk, size_t N)
{
	ImageState img;
	obtain(chunk, img.accum_alpha, N * 4, 128);
	obtain(chunk, img.center_depth, N, 128);
	obtain(chunk, img.center_alphas, N, 128);
	obtain(chunk, img.n_contrib, N * 2, 128);
	obtain(chunk, img.ranges, N, 128);
	obtain(chunk, img.point_ranges, N, 128);
	return img;
}

CudaRasterizer::BinningState CudaRasterizer::BinningState::fromChunk(char *&chunk, size_t P)
{
	BinningState binning;
	obtain(chunk, binning.point_list, P, 128);
	obtain(chunk, binning.point_list_unsorted, P, 128);
	obtain(chunk, binning.point_list_keys, P, 128);
	obtain(chunk, binning.point_list_keys_unsorted, P, 128);
	hipcub::DeviceRadixSort::SortPairs(
		nullptr, binning.sorting_size,
		binning.point_list_keys_unsorted, binning.point_list_keys,
		binning.point_list_unsorted, binning.point_list, P);
	obtain(chunk, binning.list_sorting_space, binning.sorting_size, 128);
	return binning;
}

void CudaRasterizer::Rasterizer::test(int *y)
{
	FORWARD::test(y);
}

// Forward rendering procedure for differentiable rasterization
// of Gaussians.
int CudaRasterizer::Rasterizer::forward(
	std::function<char *(size_t)> geometryBuffer,
	std::function<char *(size_t)> binningBuffer,
	std::function<char *(size_t)> imageBuffer,
	const int P, int D, int M,
	const float *background,
	const int width, int height,
	const float *means3D,
	const float *shs,
	const float *colors_precomp,
	const float *opacities,
	const float *scales,
	const float scale_modifier,
	const float *rotations,
	const int *gaussian_index,
	const float *cov3D_precomp,
	const float *view2gaussian_precomp,
	const float *viewmatrix,
	const float *projmatrix,
	const float *cam_pos,
	const float tan_fovx, float tan_fovy,
	const float kernel_size,
	const float *subpixel_offset,
	const bool prefiltered,
	bool *output,
	int *radii,
	bool debug)
{
	const float focal_y = height / (2.0f * tan_fovy);
	const float focal_x = width / (2.0f * tan_fovx);

	size_t chunk_size = required<GeometryState>(P);
	char *chunkptr = geometryBuffer(chunk_size);
	GeometryState geomState = GeometryState::fromChunk(chunkptr, P);

	if (radii == nullptr)
	{
		radii = geomState.internal_radii;
	}

	dim3 tile_grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	dim3 block(BLOCK_X, BLOCK_Y, 1);

	// Dynamically resize image-based auxiliary buffers during training
	size_t img_chunk_size = required<ImageState>(width * height);
	char *img_chunkptr = imageBuffer(img_chunk_size);
	ImageState imgState = ImageState::fromChunk(img_chunkptr, width * height);

	if (NUM_CHANNELS != 3 && colors_precomp == nullptr)
	{
		throw std::runtime_error("For non-RGB, provide precomputed Gaussian colors!");
	}

	// Allocate and intialize host array
	float *hArea = (float *)malloc(P * sizeof(float));
	memset(hArea, 0, P * sizeof(float));

	// Allocate device array
	float *dArea;
	hipMalloc(&dArea, P * sizeof(float));
	hipMemcpy(dArea, hArea, P * sizeof(float), hipMemcpyHostToDevice);

	// Run preprocessing per-Gaussian (transformation, bounding, conversion of SHs to RGB)
	CHECK_CUDA(FORWARD::preprocess(
				   P, D, M,
				   means3D,
				   (glm::vec3 *)scales,
				   scale_modifier,
				   (glm::vec4 *)rotations,
				   opacities,
				   shs,
				   geomState.clamped,
				   cov3D_precomp,
				   colors_precomp,
				   view2gaussian_precomp,
				   viewmatrix, projmatrix,
				   (glm::vec3 *)cam_pos,
				   width, height,
				   focal_x, focal_y,
				   tan_fovx, tan_fovy,
				   kernel_size,
				   radii,
				   geomState.means2D,
				   geomState.depths,
				   geomState.cov3D,
				   geomState.view2gaussian,
				   geomState.rgb,
				   geomState.conic_opacity,
				   tile_grid,
				   geomState.tiles_touched,
				   prefiltered, dArea),
			   debug)

	// Compute prefix sum over full list of touched tile counts by Gaussians
	// E.g., [2, 3, 0, 2, 1] -> [2, 5, 5, 7, 8]
	CHECK_CUDA(hipcub::DeviceScan::InclusiveSum(geomState.scanning_space, geomState.scan_size, geomState.tiles_touched, geomState.point_offsets, P), debug)

	// Retrieve total number of Gaussian instances to launch and resize aux buffers
	int num_rendered;
	CHECK_CUDA(hipMemcpy(&num_rendered, geomState.point_offsets + P - 1, sizeof(int), hipMemcpyDeviceToHost), debug);

	size_t binning_chunk_size = required<BinningState>(num_rendered);
	char *binning_chunkptr = binningBuffer(binning_chunk_size);
	BinningState binningState = BinningState::fromChunk(binning_chunkptr, num_rendered);

	// For each instance to be rendered, produce adequate [ tile | depth ] key
	// and corresponding dublicated Gaussian indices to be sorted
	duplicateWithKeys<<<(P + 255) / 256, 256>>>(
		P,
		geomState.means2D,
		geomState.depths,
		geomState.point_offsets,
		binningState.point_list_keys_unsorted,
		binningState.point_list_unsorted,
		radii,
		tile_grid)
		CHECK_CUDA(, debug)

			int bit = getHigherMsb(tile_grid.x * tile_grid.y);

	// Sort complete list of (duplicated) Gaussian indices by keys
	CHECK_CUDA(hipcub::DeviceRadixSort::SortPairs(
				   binningState.list_sorting_space,
				   binningState.sorting_size,
				   binningState.point_list_keys_unsorted, binningState.point_list_keys,
				   binningState.point_list_unsorted, binningState.point_list,
				   num_rendered, 0, 32 + bit),
			   debug)

	CHECK_CUDA(hipMemset(imgState.ranges, 0, tile_grid.x * tile_grid.y * sizeof(uint2)), debug);

	// Identify start and end of per-tile workloads in sorted list
	if (num_rendered > 0)
		identifyTileRanges<<<(num_rendered + 255) / 256, 256>>>(
			num_rendered,
			binningState.point_list_keys,
			imgState.ranges);
	CHECK_CUDA(, debug)

	// printf("in CudaRasterizer::Rasterizer::forward, P: %d num_rendered: %d geo_chunk_size: %d img_chunk_size: %d, binning_chunk_size: %d\n", P, num_rendered, chunk_size, img_chunk_size, binning_chunk_size);
	//  Let each tile blend its range of Gaussians independently in parallel
	const float *feature_ptr = colors_precomp != nullptr ? colors_precomp : geomState.rgb;
	// const float* cov3Ds = cov3D_precomp != nullptr ? cov3D_precomp : geomState.cov3D;
	const float *view2gaussian = view2gaussian_precomp != nullptr ? view2gaussian_precomp : geomState.view2gaussian;
	// const float* view2gaussian = view2gaussian_precomp;

	bool* sky_mask = NULL;
	CHECK_CUDA(FORWARD::skycull(
		tile_grid, block,
		width, height,
		sky_mask,
		focal_x, focal_y,
		imgState.ranges,
		binningState.point_list,
		view2gaussian,
		(int *)gaussian_index,
		(float3 *)scales,
		geomState.conic_opacity,
		output
	), debug);

	return num_rendered;
}